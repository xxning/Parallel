/** \file
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <errno.h>
#include <time.h>
#include <sys/time.h>
#include <stdint.h>

#define BLOCKLEN 512

int nextPower(int);
void die(const char *);
void warn(const char *);
void read_from_file(int *, char *, int);
void write_to_file(int *, char *, int);

/**
 * play - Plays the game for one step.
 * First, counts the neighbors, taking into account boundary conditions
 * Then, acts on the rules.
 * Updates need to happen all together, so a temporary new array is allocated
 */
__global__ void play(int *X, int *d_new, int N){
    //a block of threads works on a row
    int glob_i = blockIdx.x; //which row
    int temp_i = glob_i;

    //shared memory size = 3 lines * N cells each
    extern __shared__ int localArray[];

    //read row above (i-1) and write to localArray[0][j]
    if (glob_i == 0){
        temp_i = N - 1; //cyclic boundary condition
    }else{
        temp_i = glob_i - 1; //simple case
    }
    for(int j = threadIdx.x; j < N; j+=blockDim.x){
        if(j<N){
            localArray[0*N + j] = X[(temp_i)*N + j];
        }
    }

    //read own row (i) and write to localArray[1][j]
    for(int j = threadIdx.x; j < N; j+=blockDim.x){
        if(j<N){
            localArray[1*N + j] = X[(glob_i)*N + j];
        }
    }

    //read from row below (i+1) and write to localArray[2][j]
    if(glob_i == N-1){
        temp_i = 0; //cyclic boundary condition
    }else{
        temp_i = glob_i + 1; //simple case
    }
    for(int j = threadIdx.x; j < N; j+=blockDim.x){
        if(j<N){
            localArray[2*N + j] = X[(temp_i)*N + j];
        }
    }
    //wait for shared memory to be "full"
    __syncthreads();

    //shared memory is now complete, we're ready to operate on it
    int up, down, left, right;
    for(int j = threadIdx.x; j < N; j+=blockDim.x){
        if (j < N){
            int idx = N*glob_i + j;

            up = 0;
            down = 2;
            //cyclic boundary conditions
            left =  j == 0 ? N - 1 : j - 1;
            right = j == N-1 ? 0 : j + 1; 

            int sum = 
                localArray[N*up+left]+       //i-1, j-1
                localArray[N*up+j]+          //i-1, j
                localArray[N*up+right]+      //i-1, j+1

                localArray[N*1+left]+        //i, j-1
                localArray[N*1+right]+       //i, j+1

                localArray[N*down+left]+     //i+1, j-1
                localArray[N*down+j]+        //i+1, j
                localArray[N*down+right];    //i+1, j+1

            
            //act based on rules - write to global array
            if(localArray[1*N + j] == 0  && sum == 3 ){
                d_new[idx]=1; //born
            }else if ( localArray[1*N + j] == 1  && (sum < 2 || sum>3 ) ){
                d_new[idx]=0; //dies - loneliness or overpopulation
            }else{
                d_new[idx] = localArray[1*N + j]; //nothing changes
            }
        }
    }
    return;
}

/**
 * main - plays the game of life for t steps according to the rules:
 * - A dead(0) cell with exactly 3 living neighbors becomes alive (birth)
 * - A dead(0) cell with any other number of neighbors stays dead (barren)
 * - A live(1) cell with 0 or 1 living neighbors dies (loneliness)
 * - A live(1) cell with 4 or more living neighbors dies (overpopulation)
 * - A live(1) cell with 2 or 3 living neighbors stays alive (survival)
 */
int main(int argc, char **argv){

    //sanity check for input
    if(argc !=4){
        printf("Usage: %s filename size t, where:\n", argv[0]);
        printf("\tfilename is the input file \n");
        printf("\tsize is the grid side and \n");
        printf("\tt generations to play\n");
        die("Wrong arguments");
    }

    //declarations
    char *filename = argv[1];
    int N = atoi(argv[2]);
    int t = atoi(argv[3]);
    int gen = 0;
    int *table = (int *)malloc(N*N*sizeof(int));
    if (!table)
        die("Couldn't allocate memory to table");

    //CUDA - divide the table in N blocks of 1 line, 512 threads per block
    dim3 threadsPerBlock(BLOCKLEN, 1); //max threads/block
    dim3 numBlocks(N, 1); //split board into blocks

    //CUDA - timing
    float gputime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //read input
    read_from_file(table, filename, N);

    //CUDA - copy to device
    int *d_table;
    hipMalloc(&d_table, N*N*sizeof(int));
    int *d_new;
    hipMalloc(&d_new, N*N*sizeof(int));
    hipEventRecord(start, 0);
    hipMemcpy(d_table, table, N*N*sizeof(int), hipMemcpyHostToDevice);

    //play game for t generations
    for(gen=0; gen<t; gen++){

        //alternate between using d_table and d_new as temp
        if(gen%2==0){
            //3*N*sizeof(int): size for shared memory
            play<<<numBlocks, threadsPerBlock, 3*N*sizeof(int)>>>(d_table /*data*/, d_new /*temp*/, N);
        }else{
            play<<<numBlocks, threadsPerBlock, 3*N*sizeof(int)>>>(d_new /*data*/, d_table /*temp*/, N);
        }
        hipDeviceSynchronize(); //don't continue if kernel not done

    }
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&gputime, start, stop);
    printf("[%d]\t %g \n",gen, gputime/1000.0f);

    //copy back from device
    if(t%2==1){
        hipMemcpy(table, d_new, N*N*sizeof(int), hipMemcpyDeviceToHost);
    }else{
        hipMemcpy(table, d_table, N*N*sizeof(int), hipMemcpyDeviceToHost);
    }
    //save output for later
    write_to_file(table, filename, N);

    free(table);
    hipFree(d_new);
    hipFree(d_table);
    return 0;
}

/**
 * die - display an error and terminate.
 * Used when some fatal error happens
 * and continuing would mess things up.
 */
void die(const char *message){
    if(errno){
        perror(message);
    }else{
        printf("Error: %s\n", message);
    }
    exit(1);
}

/**
 * warn - display a warning and continue
 * used when something didn't go as expected
 */
void warn(const char *message){
    if(errno){
        perror(message);
    }else{
        printf("Warning: %s\n", message);
    }
    return;
}

/**
 * read_from_file - read N*N integer values from an appropriate file.
 * Saves the game's board into array X for use by other functions
 * Warns or kills the program if something goes wrong
 */
void read_from_file(int *X, char *filename, int N){

    FILE *fp = fopen(filename, "r+");
    int size = fread(X, sizeof(int), N*N, fp);
    if(!fp)
        die("Couldn't open file to read");
    if(!size)
        die("Couldn't read from file");
    if(N*N != size)
        warn("Expected to read different number of elements");

    printf("elements read: %d\n", size);

    fclose(fp);
    return;
}

/**
 * write_to_file - write N*N integer values to a binary file.
 * Saves game's board from array X to the file
 * Names the file tableNxN_new.bin, so the input file is not overwritten
 */
void write_to_file(int *X, char *filename, int N){

    //save as tableNxN_new.bin
    char newfilename[100];
    sprintf(newfilename, "cuda_table%dx%d.bin", N, N);
    printf("writing to: %s\n", newfilename);

    FILE *fp;
    int size;
    if( ! ( fp = fopen(newfilename, "w+") ) )
        die("Couldn't open file to write");
    if( ! (size = fwrite(X, sizeof(int), N*N, fp)) )
        die("Couldn't write to file");
    if (size != N*N)
        warn("Expected to write different number of elements");

    fclose(fp);
    return;
}

int nextPower(int N){
    int n=0;
    while(1){
        if(1<<n < N){
            n++;
        }else{
            return 1<<n;
        }
    }
}
